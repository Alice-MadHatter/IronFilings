#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <>

#include <thrust\host_vector.h>
#include <thrust\device_vector.h>

#include <glfw\glfw3.h>
#include <glm\glm.hpp>

#include "application.cuh"

using namespace std;
using namespace thrust;
using namespace glm;

class MyData
{
public:
	float scalar;
	vec3 vector;
	__host__ __device__ MyData() : scalar(0.0f), vector(0.0f) {};
};

__global__ void myFunc(MyData* data, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size)
		data[index].vector *= data[index].scalar;
}

int main(int argc, char** argv)
{
	/************************************* CUDA Test Begin *************************************/

	// vector size
	const int vecSize = 65536;

	// initialize vectors
	host_vector<MyData> hostVec(vecSize);
	device_vector<MyData> deviceVec(vecSize);
	for (int i = 0; i < vecSize; i++)
	{
		hostVec[i].vector.x = i + 0; hostVec[i].vector.y = i + 1; hostVec[i].vector.z = i + 2;
		hostVec[i].scalar = i;
	}

	// get device property
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int maxThreadNumber = deviceProp.maxThreadsDim[0];
	int maxBlockNumber = deviceProp.maxGridSize[0];

	// memory copy & calculation
	hipMemcpy(raw_pointer_cast(deviceVec.data()), raw_pointer_cast(hostVec.data()), sizeof(vec3) * hostVec.size(), hipMemcpyHostToDevice);
	myFunc << <maxBlockNumber, maxThreadNumber >> >(raw_pointer_cast(deviceVec.data()), vecSize);
	hipMemcpy(raw_pointer_cast(hostVec.data()), raw_pointer_cast(deviceVec.data()), sizeof(vec3) * deviceVec.size(), hipMemcpyDeviceToHost);

	// print partial result
	cout << "Hello IronFilings !" << endl;
	for (int i = 0; i < 100; i++)
		cout << "(" << hostVec[i].vector.x << ", " << hostVec[i].vector.y << ", " << hostVec[i].vector.z << ") ";
	cout << endl;

	/************************************* CUDA Test End *************************************/

	/************************************* GLFW Test Begin *************************************/

	Application app("Hello Iron Filings!", 1024, 768, false);
	app.run();

	/************************************* GLFW Test End *************************************/

	return EXIT_SUCCESS;
}
